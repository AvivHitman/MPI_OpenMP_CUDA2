#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "definition.h"
#include <string.h>

__global__  void getTheScore(int *scores, char* dna, char* rna, int numElements);
__device__ void compare(int *scores, char dna, char rna);
__device__ int checkColon(char a, char b);
__device__ int checkPoint(char a, char b);
__global__  void arrayOfScores(double *s, char* dna, char* rna,double* weight, int numElements);
__device__ void compare2 (double *s, char dna, char rna, double* weight, int i);


__global__  void getTheScore(int *scores, char* dna, char* rna, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	
    // compare dna & rna in place i by thread i 
    if (i < numElements){
		compare(scores, dna[i], rna[i]);
	}
	
        
}

__global__  void arrayOfScores(double *s, char* dna, char* rna, double* weight, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	
    // compare dna & rna in place i by thread i and save each reslt in array
    if (i < numElements){
		compare2(s, dna[i], rna[i], weight, i);

	}
	
        
}
// create array in size rna 
__device__ void compare2 (double *s, char dna, char rna, double* weight, int i){

		if (dna == rna)
			s[i] = weight[0]; //num of stars
		
		else if (checkColon(dna, rna) == 1)
			s[i] = -weight[1]; // num of colons

		else if (checkPoint(dna, rna) == 1)
			s[i] = -weight[2]; // num of points

		else
			s[i] = -weight[3]; // num of spaces


	

}
// create array in size 4 that we can get from it the total score quickly
__device__ void compare (int *scores, char dna, char rna){
		for(int k=0; k<4; k++){
			scores[k] = 0;
		}

		if (dna == rna)
			atomicAdd(&scores[0],1); //num of stars
		
		else if (checkColon(dna, rna) == 1)
			atomicAdd(&scores[1],1); // num of colons

		else if (checkPoint(dna, rna) == 1)
			atomicAdd(&scores[2],1); // num of points

		else
			atomicAdd(&scores[3],1); // num of spaces


	

}
__device__ int checkColon(char a, char b) {
	int flag;
	int k;
	const char *conserativeGroup[9] = { "NDEQ", "MILV", "FYM", "NEQK", "QHRK",
			"HY", "STA", "NHQK", "MILF" };
	for (int j = 0; j < 9; j++) {
		flag = 0;
		k = 0;
		while (conserativeGroup[j][k] != '\0') {
			if ((conserativeGroup[j][k] == a)
					|| (conserativeGroup[j][k] == b)) {
				flag++;
			}
			k++;

		}
		if (flag == 2)
			return 1;


	}
	return 0;
}

__device__ int checkPoint(char a, char b) {
	int flag;
	int k;
	const char *semiConserativeGroup[11] = { "SAG", "SGND", "NEQHRK", "ATV",
			"STPA", "NDEQHK", "HFY", "CSA", "STNK", "SNDEQK", "FVLIM" };
	for (int j = 0; j < 11; j++) {
		flag = 0;
		k = 0;
		while (semiConserativeGroup[j][k] != '\0') {
			if ((semiConserativeGroup[j][k] == a)
					|| (semiConserativeGroup[j][k] == b)) {
				flag++;
			}
			k++;

		}
		if (flag == 2)
			return 1;
	}

	return 0;
}

int computeOnGPU(int *scores, char* dna, char* rna, int n) {
	
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

  
    // Allocate memory on GPU to copy the data from the host --> dna
    char *d_A;
    err = hipMalloc((void **)&d_A, strlen(dna)*sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   // Allocate memory on GPU to copy the data from the host--> rna
    char *d_B;
    err = hipMalloc((void **)&d_B, strlen(rna)*sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   // Allocate memory on GPU to copy the data from the host--> scores
    int *d_C;
    err = hipMalloc((void **)&d_C, 4*sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory --> dna
    err = hipMemcpy(d_A, dna, strlen(dna)*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }	 


  // Copy data from host to the GPU memory--> rna
    err = hipMemcpy(d_B, rna, strlen(rna)*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	

    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(strlen(rna) + threadsPerBlock - 1) / threadsPerBlock;
    getTheScore<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_A, d_B, strlen(rna));
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	

    // Copy the  result from GPU to the host memory--> scores
    err = hipMemcpy(scores, d_C, 4*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	
    // Free allocated memory on GPU
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (hipFree(d_B) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (hipFree(d_C) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}




int computeOnGPU2(double *s, char* dna, char* rna, double* weight, int n) {
	
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

  
    // Allocate memory on GPU to copy the data from the host --> dna
    char *d_A;
    err = hipMalloc((void **)&d_A, strlen(dna)*sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   // Allocate memory on GPU to copy the data from the host--> rna
    char *d_B;
    err = hipMalloc((void **)&d_B, strlen(rna)*sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   // Allocate memory on GPU to copy the data from the host--> scores
    double *d_C;
    err = hipMalloc((void **)&d_C, strlen(rna)*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

// Allocate memory on GPU to copy the data from the host--> weight
    double *d_D;
    err = hipMalloc((void **)&d_D, 4*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory--> dna
    err = hipMemcpy(d_A, dna, strlen(dna)*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }	 


  // Copy data from host to the GPU memory-->rna
    err = hipMemcpy(d_B, rna, strlen(rna)*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

// Copy data from host to the GPU memory--> weight
    err = hipMemcpy(d_D, weight, 4*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	

    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(strlen(rna) + threadsPerBlock - 1) / threadsPerBlock;
    arrayOfScores<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_A, d_B, d_D, strlen(rna));
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	

    // Copy the  result from GPU to the host memory --> scores
    err = hipMemcpy(s, d_C, strlen(rna)*sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	
    // Free allocated memory on GPU
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (hipFree(d_B) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (hipFree(d_C) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (hipFree(d_D) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

